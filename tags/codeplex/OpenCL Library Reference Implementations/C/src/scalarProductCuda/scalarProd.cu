#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample calculates scalar products of a 
 * given set of input vector pairs
 */

//Sources:
// [1] http://developer.nvidia.com/cuda-cc-sdk-code-samples --> sample: Scalar Product

// Usage:
// Command line parameters:
	// n : number of input elements per Vector
	// w : number of warmup rounds
	// m : number of measured rounds
	// c : number of cooldown rounds
	
	// example: -m=10 => 10 rounds with measured time are executed

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <cutil_inline.h>
#include <shrQATest.h>


///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C"
void scalarProdCPU(
    float *h_C,
    float *h_A,
    float *h_B,
    int vectorN,
    int elementN
);



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cu"



////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////

//Total number of input vector pairs; arbitrary
const int VECTOR_N = 1;
//Number of elements per vector; arbitrary, 
//but strongly preferred to be a multiple of warp size
//to meet memory coalescing constraints
int ELEMENT_N = 4096;
//Total number of data elements
int    DATA_N = VECTOR_N * ELEMENT_N;

int   DATA_SZ = DATA_N * sizeof(float);
int RESULT_SZ = VECTOR_N  * sizeof(float);

int numWarmupRounds = 5;
int numMeasuredRounds = 3;
int numCoolDownRounds = 2;


void processCommandLine(int argc, char **argv)
{
	int cmdVali = 0;
	if (cutGetCmdLineArgumenti( argc, (const char**)argv, "n", &cmdVali))
	{
		ELEMENT_N = cmdVali;
		printf("The number of elements is set to: %d\n", ELEMENT_N);
		DATA_N = VECTOR_N * ELEMENT_N;
		DATA_SZ = DATA_N * sizeof(float);
		RESULT_SZ = VECTOR_N  * sizeof(float);
	}
	if (cutGetCmdLineArgumenti(argc, (const char**)argv, "w", &cmdVali))
	{
		numWarmupRounds = cmdVali;
		printf("The number of warm-up rounds is set to: %d\n", numWarmupRounds);
	}
	if (cutGetCmdLineArgumenti(argc, (const char**)argv, "m", &cmdVali))
	{
		numMeasuredRounds = cmdVali;
		printf("The number of warm-up rounds is set to: %d\n", numMeasuredRounds);
	} 
	if (cutGetCmdLineArgumenti(argc, (const char**)argv, "c", &cmdVali))
	{
		numCoolDownRounds = cmdVali;
		printf("The number of warm-up rounds is set to: %d\n", numCoolDownRounds);
	}
}



///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C_CPU, *h_C_GPU;
    float *d_A, *d_B, *d_C;
    double delta, ref, sum_delta, sum_ref, L1norm;
    unsigned int hTimer;
    int i;

    shrQAStart(argc, argv);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

	processCommandLine(argc, argv);

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Initializing data...\n");
        printf("...allocating CPU memory.\n");
        h_A     = (float *)malloc(DATA_SZ);
        h_B     = (float *)malloc(DATA_SZ);
        h_C_CPU = (float *)malloc(RESULT_SZ);
        h_C_GPU = (float *)malloc(RESULT_SZ);

        printf("...allocating GPU memory.\n");
        cutilSafeCall( hipMalloc((void **)&d_A, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_B, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_C, RESULT_SZ) );

        printf("...generating input data in CPU mem.\n");
        srand(123);
        //Generating input data on CPU
        for(i = 0; i < DATA_N; i++){
            h_A[i] = RandFloat(0.0f, 1.0f);
            h_B[i] = RandFloat(0.0f, 1.0f);
        }

        printf("...copying input data to GPU mem.\n");
        //Copy options data to GPU memory for further processing 
        cutilSafeCall( hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice) );
		printf("Data init done.\n");


		printf("Executing GPU kernel...\n");

		for (int i = 0; i < numWarmupRounds; i++)
		{
			scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
			cutilCheckMsg("scalarProdGPU() execution failed\n");
		}

		float overallTime = 0.0f;
		for (int i = 0; i < numMeasuredRounds; i++)
		{
			cutilSafeCall( cutilDeviceSynchronize() );
			cutilCheckError( cutResetTimer(hTimer) );
			cutilCheckError( cutStartTimer(hTimer) );
			scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
			cutilCheckMsg("scalarProdGPU() execution failed\n");
			cutilSafeCall( cutilDeviceSynchronize() );
			cutilCheckError( cutStopTimer(hTimer) );
			float time = cutGetTimerValue(hTimer);
			overallTime += time;
			printf("The elapsed time for round %d is: %f ms.\n", i+1, time);
		}

		for(int i = 0; i < numCoolDownRounds; i++)
		{
			scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
			cutilCheckMsg("scalarProdGPU() execution failed\n");
		}

		overallTime /= numMeasuredRounds;	
	printf("The average elapsed time is: %f ms.\n\n", overallTime);

    printf("Reading back GPU result...\n");
        //Read back GPU results to compare them to CPU results
        cutilSafeCall( hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost) );


    printf("Checking GPU results...\n");
        printf("..running CPU scalar product calculation\n");
        scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);

        printf("...comparing the results\n");
        //Calculate max absolute difference and L1 distance
        //between CPU and GPU results
        sum_delta = 0;
        sum_ref   = 0;
        for(i = 0; i < VECTOR_N; i++){
            delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
            ref   = h_C_CPU[i];
            sum_delta += delta;
            sum_ref   += ref;
        }
        L1norm = sum_delta / sum_ref;

    printf("Shutting down...\n");
        cutilSafeCall( hipFree(d_C) );
        cutilSafeCall( hipFree(d_B)   );
        cutilSafeCall( hipFree(d_A)   );
        free(h_C_GPU);
        free(h_C_CPU);
        free(h_B);
        free(h_A);
        cutilCheckError( cutDeleteTimer(hTimer) );

    cutilDeviceReset();
    printf("L1 error: %E\n", L1norm);
    shrQAFinishExit(argc, (const char **)argv, (L1norm < 1e-6) ? QA_PASSED : QA_FAILED);
}
